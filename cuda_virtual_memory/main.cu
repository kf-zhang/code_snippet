#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <vector>
#include <iostream>
#include <map>
#include <optional>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

void check(hipError_t err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        const char* errorString;
        hipDrvGetErrorString(err, &errorString);
        std::cerr << "CUDA error at " << file << ":" << line << " code=" << static_cast<unsigned int>(err)
                  << " \"" << errorString << "\" " << func << std::endl;
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

/***********warpper for cuda api***********/
bool virtualAddressSupported(int device) {
    int virtualAddressing = 0;
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&virtualAddressing, CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED, device));
    return virtualAddressing != 0;
}

hipDeviceptr_t allocateVirtualAddress(size_t size, size_t alignment = 0) {
    // Reserve a virtual address range with the specified size and alignment
    // When alignment is set to 0, the default alignment is used
    hipDeviceptr_t d_ptr;
    CHECK_CUDA_ERROR(hipMemAddressReserve(&d_ptr, size, alignment, 0, 0));
    return d_ptr;
}

void freeVirtualAddress(hipDeviceptr_t d_ptr, size_t size) {
    // Free the virtual address range
    CHECK_CUDA_ERROR(hipMemAddressFree(d_ptr, size));
}

size_t getMemoryGranularity(int device) {
    // Get the memory granularity of the device
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;
    size_t granularity;
    CHECK_CUDA_ERROR(hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum));
    return granularity;
}

hipMemGenericAllocationHandle_t allocatePhysicalMemory(size_t size, int device) {
    // Allocate physical memory on the specified device
    hipMemGenericAllocationHandle_t handle;
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;
    
    size_t granularity = getMemoryGranularity(device);
    assert(size % granularity == 0);
    CHECK_CUDA_ERROR(hipMemCreate(&handle, size, &prop, 0));
    return handle;
}

void freePhysicalMemory(hipMemGenericAllocationHandle_t handle) {
    // Free the physical memory
    CHECK_CUDA_ERROR(hipMemRelease(handle));
}

//map physical memory to the allocated virtual address
void mapPA2VA(hipDeviceptr_t d_ptr, size_t size, hipMemGenericAllocationHandle_t handle, int device, int offset) {
    // Map physical memory to the reserved virtual address
    // virtual_memory[d_ptr, d_ptr + size) = physical_memory[offset, offset + size)
    // The handle is the handle to the physical allocation
    // The device is the device where the physical allocation resides
    CHECK_CUDA_ERROR(hipMemMap(d_ptr, size, offset, handle, 0));

    // Set access flags for the mapped memory
    hipMemAccessDesc accessDesc;
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = device;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    CHECK_CUDA_ERROR(hipMemSetAccess(d_ptr, size, &accessDesc, 1));
}

void unmapPA2VA(hipDeviceptr_t d_ptr, size_t size) {
    // Unmap the physical memory
    CHECK_CUDA_ERROR(hipMemUnmap(d_ptr, size));
}
/***********warpper for cuda api***********/

int main() {
            hipCtx_t cuContext;
    hipDeviceptr_t d_ptr;
    int device = 0;
    size_t size = (1<<24) * sizeof(int); 

    // Initialize CUDA driver API
    CHECK_CUDA_ERROR(hipInit(0));
    hipDevice_t cuDevice;
    CHECK_CUDA_ERROR(hipDeviceGet(&cuDevice, device));
    CHECK_CUDA_ERROR(hipCtxCreate(&cuContext, 0, cuDevice));

    // Check if the device supports virtual address management
    if (!virtualAddressSupported(device)) {
        std::cerr << "Virtual address management is not supported on device " << device << std::endl;
        return;
    }
    // Reserve a virtual address range
    ////the minimum size of the virtual address range is 2MB, set alignment to 0 means the default alignment
    d_ptr = allocateVirtualAddress(size, 0);


    //request the allocation granularity of the device
    hipMemGenericAllocationHandle_t handle = allocatePhysicalMemory(size, device);

    // Map the physical memory to the reserved virtual address
    mapPA2VA(d_ptr, size, handle, device, 0);

    // Use the mapped memory
    int num_elements = size / sizeof(int);
    int* h_ptr = new int[num_elements];
    for (size_t i = 0; i < num_elements; ++i) {
        h_ptr[i] = static_cast<int>(i);
    }
    CHECK_CUDA_ERROR(hipMemcpyHtoD(d_ptr, h_ptr, size));

    // Print some values from the mapped memory
    int* h_ptr_out = new int[num_elements];
    CHECK_CUDA_ERROR(hipMemcpyDtoH(h_ptr_out, d_ptr, size));
    bool success = true;
    for (size_t i = 0; i < num_elements; ++i) {
        if (h_ptr_out[i] != h_ptr[i]) {
            std::cerr << "Mismatch at index " << i << ": " << h_ptr_out[i] << " != " << h_ptr[i] << std::endl;
            success = false;
            break;
        }
    }
    if (success) {
        std::cout << "All values match" << std::endl;
    }

    // Unmap and release resources
    unmapPA2VA(d_ptr, size);
    freePhysicalMemory(handle);
    freeVirtualAddress(d_ptr, size);

    // Clean up
    delete[] h_ptr;
    delete[] h_ptr_out;
    CHECK_CUDA_ERROR(hipCtxDestroy(cuContext));
}